#include "hip/hip_runtime.h"
/*
 * This is a modified version of the diff-gaussian-rasterization.
 * The original license still applies, see the original copyright notice below:
 *
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_sampler/config.h"
#include "cuda_sampler/sampler.h"
#include <fstream>
#include <string>

std::function<char*(size_t N)> resize_functional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
        return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
SampleGaussiansCUDA(
    const torch::Tensor& means,
    const torch::Tensor& values,
    const torch::Tensor& covariances,
    const torch::Tensor& conics,
    const torch::Tensor& opacities,
    const torch::Tensor& samples,
    const bool debug)
{
    const int P = means.size(0);
    const int D = means.size(-1);
    const int N = samples.size(0);
    const int C = values.size(-1);

    auto int_opts = means.options().dtype(torch::kInt32);
    auto float_opts = means.options().dtype(means.dtype());

    torch::Tensor radii = torch::full({P}, 0, float_opts);
    torch::Tensor out_values = torch::full({N, C}, 0.0, float_opts);

    torch::Device device(torch::kCUDA);
    torch::TensorOptions options(torch::kByte);
    torch::Tensor geom_buffer = torch::empty({0}, options.device(device));
    torch::Tensor binning_buffer = torch::empty({0}, options.device(device));
    torch::Tensor sample_binning_buffer = torch::empty({0}, options.device(device));
    std::function<char*(size_t)> geom_func = resize_functional(geom_buffer);
    std::function<char*(size_t)> binning_func = resize_functional(binning_buffer);
    std::function<char*(size_t)> sample_binning_func = resize_functional(sample_binning_buffer);

    const torch::Tensor min_bound = std::get<0>(samples.min(0));
    const torch::Tensor max_bound = std::get<0>(samples.max(0));

    const torch::Tensor tile_grid = torch::ceil((max_bound - min_bound + 1e-6f) / BLOCK_SIZE).to(torch::kInt32);
    const int blocks = torch::prod(tile_grid).item<int>();

    torch::Tensor ranges = torch::full({blocks * (long) sizeof(uint2) + 8}, 0, options.device(device));
    torch::Tensor sample_ranges = torch::full({blocks * (long) sizeof(uint2) + 8}, 0, options.device(device));

    int rendered = 0;
    if (P != 0) {
        rendered = CudaSampler::Sampler::preprocess(
          geom_func,
          binning_func,
          sample_binning_func,
          P, D, N, C, blocks,
          tile_grid.contiguous().data<int>(),
          min_bound.contiguous().data<FLOAT>(),
          means.contiguous().data<FLOAT>(),
          values.contiguous().data<FLOAT>(),
          covariances.contiguous().data<FLOAT>(),
          conics.contiguous().data<FLOAT>(),
          opacities.contiguous().data<FLOAT>(),
          samples.contiguous().data<FLOAT>(),
          reinterpret_cast<uint2*>(ranges.contiguous().data_ptr()),
          reinterpret_cast<uint2*>(sample_ranges.contiguous().data_ptr()),
          radii.contiguous().data<FLOAT>(),
          debug);
    }

    CudaSampler::Sampler::forward(
        P, D, N, C, blocks, rendered,
        means.contiguous().data<FLOAT>(),
        values.contiguous().data<FLOAT>(),
        conics.contiguous().data<FLOAT>(),
        opacities.contiguous().data<FLOAT>(),
        samples.contiguous().data<FLOAT>(),
        reinterpret_cast<char*>(binning_buffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(sample_binning_buffer.contiguous().data_ptr()),
        reinterpret_cast<uint2*>(ranges.contiguous().data_ptr()),
        reinterpret_cast<uint2*>(sample_ranges.contiguous().data_ptr()),
        radii.contiguous().data<FLOAT>(),
        out_values.contiguous().data<FLOAT>(),
        debug);

    return std::make_tuple(rendered, out_values, binning_buffer, sample_binning_buffer, ranges, sample_ranges);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
SampleGaussiansBackwardCUDA(
    const torch::Tensor& means,
    const torch::Tensor& values,
    const torch::Tensor& conics,
    const torch::Tensor& opacities,
    const torch::Tensor& samples,
    const int num_rendered,
    const torch::Tensor& dL,
	const torch::Tensor& binning_buffer,
	const torch::Tensor& sample_binning_buffer,
    const torch::Tensor& ranges,
	const torch::Tensor& sample_ranges,
    const bool debug)
{
    const int P = means.size(0);
    const int D = means.size(-1);
    const int N = samples.size(0);
    const int C = values.size(-1);

    torch::Tensor dL_dmeans = torch::zeros({P, D}, means.options());
    torch::Tensor dL_dvalues = torch::zeros({P, C}, means.options());
    torch::Tensor dL_dconics = torch::zeros({P, D, D}, means.options());
    torch::Tensor dL_dopacities = torch::zeros({P}, means.options());
    torch::Tensor dL_dsamples = torch::zeros({N, D}, means.options());

    const torch::Tensor min_bound = std::get<0>(samples.min(0));
    const torch::Tensor max_bound = std::get<0>(samples.max(0));

    const torch::Tensor tile_grid = torch::ceil((max_bound - min_bound + 1e-6f) / BLOCK_SIZE).to(torch::kInt32);
    const int blocks = torch::prod(tile_grid).item<int>();

    if (P != 0) {
        CudaSampler::Sampler::backward(
            P, D, N, C, blocks, num_rendered,
            means.contiguous().data<FLOAT>(),
            values.contiguous().data<FLOAT>(),
            conics.contiguous().data<FLOAT>(),
            opacities.contiguous().data<FLOAT>(),
            samples.contiguous().data<FLOAT>(),
            reinterpret_cast<char*>(binning_buffer.contiguous().data_ptr()),
            reinterpret_cast<char*>(sample_binning_buffer.contiguous().data_ptr()),
            reinterpret_cast<uint2*>(ranges.contiguous().data_ptr()),
            reinterpret_cast<uint2*>(sample_ranges.contiguous().data_ptr()),
            dL.contiguous().data<FLOAT>(),
            dL_dmeans.contiguous().data<FLOAT>(),
            dL_dvalues.contiguous().data<FLOAT>(),
            dL_dconics.contiguous().data<FLOAT>(),
            dL_dopacities.contiguous().data<FLOAT>(),
            dL_dsamples.contiguous().data<FLOAT>(),
            debug);
    }

    return std::make_tuple(dL_dmeans, dL_dvalues, dL_dconics, dL_dopacities, dL_dsamples);
}
